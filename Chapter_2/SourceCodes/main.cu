#include <stdio.h>
#include <cstdio>
#include <hip/hip_runtime.h>


__global__
void vecAddKernel(float* A, float* B, float* C, int n){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i<n){
        C[i] = A[i] + B[i];
    }
}





void vecAdd(float* A_h, float *B_h, float *C_h, int n){
    int size = n* sizeof(float);
    float *A_d, *B_d, *C_d;

    hipError_t err;

    // Part 1: Allocate device memory for A, B, and C
    // Copy A and B to device memory
    err = hipMalloc((void**)&A_d, size);
    if (err == hipSuccess){
        printf("Successful device memory allocation!\n");
    }else{
        printf("Unsuccessful memory allocation!\n");
    }

    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    // Part 2.1: Copy from host to device
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n/256.0), 256>>>(A_d, B_d, C_d, n);


    // Part 2.3: Copy from device to host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);


    // Part 3: Free the allocated memory on teh device
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main(){

    float a[] = {1,2,3};
    float b[] = {3,4,5};

    float c[] = {0.0,0.0,0.0};

    int N = 3;


    vecAdd(a, b, c, N);

    for (int i=0; i<N; i++){
        printf("number: %f\n", c[i]);
    }

    return 0;
    
}